#include "hip/hip_runtime.h"
#include "util.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <algorithm>    // std::min std::max

using namespace std;

// Ej 1a) Kernel 
__global__ void transpose_global_kernel(float* d_input, int width, int height, float* d_output) {
    int imgx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int imgy = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (imgx < width && imgy < height) {
        d_output[(imgx*height) + imgy] = d_input[(imgy*width) + imgx];
    }
}

// Ej 1a) Kernel 
__global__ void transpose_shared_kernel(float* d_input, int width, int height, float* d_output) {
    
    // TODO: Tamaño constante
    __shared__ float tile[1024];

    unsigned int imgx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int imgy = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Thread (tx,ty) lee pixel (x,y) (global) y escribe pixel (tx,ty) (compartida) 
    tile[threadIdx.y * blockDim.y + threadIdx.x] = d_input[(imgy*width) + imgx];
    __syncthreads();

    // Thread (tx,ty) lee pixel (ty,tx) (compartida) y escribe pixel (y,x) (global)
    if (imgx < width && imgy < height) {
        d_output[(imgx*height) + imgy] = tile[threadIdx.y * blockDim.y + threadIdx.x];
        // Acceso por columna, no anda
        // d_output[(imgx*height) + imgy] = tile[threadIdx.x * blockDim.x + threadIdx.y];
    }
}

// Ej 1a) A
void transpose_global(float * img_in, int width, int height, float * img_out){
    
    // Auxiliar para contar tiempo total
    float t_total = 0;
    
    // Etapa 1: Reserva de Memoria
    CLK_CUEVTS_INIT;
    CLK_CUEVTS_START;
    // Reserva en CPU
    unsigned int size = width * height * sizeof(float);
    float * device_img_in = (float *)malloc(size);
    float * device_img_out = (float *)malloc(size);
    // Reserva en GPU
    CUDA_CHK(hipMalloc((void**)& device_img_in, size));
    CUDA_CHK(hipMalloc((void**)& device_img_out, size));
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion global (Reserva de memoria): %f ms\n", t_elap);
    t_total = t_total + t_elap;
    
    // Etapa 2: Transferencia de datos (Host -> Device)
    CLK_CUEVTS_START;
    CUDA_CHK(hipMemcpy(device_img_in, img_in, size, hipMemcpyHostToDevice)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion global (Transferencia de datos (Host -> Device)): %f ms\n", t_elap);
    t_total = t_total + t_elap;

    // Etapa 3: Definir grilla
    int block_size = 32; // TODO: Definir constante
    int block_amount_x = width / block_size + (width % block_size != 0); // Division with ceiling
    int block_amount_y = height / block_size + (height % block_size != 0); // Division with ceiling
    dim3 tamGrid(block_amount_x, block_amount_y); // Grid dimension
    dim3 tamBlock(block_size, block_size); // Block dimension

    // Etapa 4 : Lanzar Kernel
    CLK_CUEVTS_START;
    transpose_global_kernel<<<tamGrid, tamBlock>>>(device_img_in, width, height, device_img_out);
    // Sincronizar threads antes de parar timers
    hipDeviceSynchronize(); 
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion global (Kernel): %f ms\n", t_elap);
    t_total = t_total + t_elap;

    // Etapa 5: Transferencia de Datos (Device -> Host)
    CLK_CUEVTS_START;
    CUDA_CHK(hipMemcpy(img_out, device_img_out, size, hipMemcpyDeviceToHost)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion global (Transferencia de datos (Host <- Device)): %f ms\n", t_elap);
    t_total = t_total + t_elap;
    printf("Tiempo transposicion global: %f ms\n", t_total);
    printf("\n");

    // Etapa 6: Liberación de Memoria
    CUDA_CHK(hipFree(device_img_in));
    CUDA_CHK(hipFree(device_img_out));
}

// Ej 1b) A
void transpose_shared(float * img_in, int width, int height, float * img_out){
    
    // Auxiliar para contar tiempo total
    float t_total = 0;
    
    // Etapa 1: Reserva de Memoria
    CLK_CUEVTS_INIT;
    CLK_CUEVTS_START;
    // Reserva en CPU
    unsigned int size = width * height * sizeof(float);
    float * device_img_in = (float *)malloc(size);
    float * device_img_out = (float *)malloc(size);
    // Reserva en GPU
    CUDA_CHK(hipMalloc((void**)& device_img_in, size));
    CUDA_CHK(hipMalloc((void**)& device_img_out, size));
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion global (Reserva de memoria): %f ms\n", t_elap);
    t_total = t_total + t_elap;
    
    // Etapa 2: Transferencia de datos (Host -> Device)
    CLK_CUEVTS_START;
    CUDA_CHK(hipMemcpy(device_img_in, img_in, size, hipMemcpyHostToDevice)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion global (Transferencia de datos (Host -> Device)): %f ms\n", t_elap);
    t_total = t_total + t_elap;

    // Etapa 3: Definir grilla
    int block_size = 32; // TODO: Definir constante
    int block_amount_x = width / block_size + (width % block_size != 0); // Division with ceiling
    int block_amount_y = height / block_size + (height % block_size != 0); // Division with ceiling
    dim3 tamGrid(block_amount_x, block_amount_y); // Grid dimension
    dim3 tamBlock(block_size, block_size); // Block dimension

    // Etapa 4 : Lanzar Kernel
    CLK_CUEVTS_START;
    transpose_shared_kernel<<<tamGrid, tamBlock>>>(device_img_in, width, height, device_img_out);
    // Sincronizar threads antes de parar timers
    hipDeviceSynchronize(); 
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion global (Kernel): %f ms\n", t_elap);
    t_total = t_total + t_elap;

    // Etapa 5: Transferencia de Datos (Device -> Host)
    CLK_CUEVTS_START;
    CUDA_CHK(hipMemcpy(img_out, device_img_out, size, hipMemcpyDeviceToHost)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion global (Transferencia de datos (Host <- Device)): %f ms\n", t_elap);
    t_total = t_total + t_elap;
    printf("Tiempo transposicion global: %f ms\n", t_total);
    printf("\n");

    // Etapa 6: Liberación de Memoria
    CUDA_CHK(hipFree(device_img_in));
    CUDA_CHK(hipFree(device_img_out));
}
