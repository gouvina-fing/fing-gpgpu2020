#include "hip/hip_runtime.h"
#include "util.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <algorithm>    // std::min std::max

using namespace std;

#define TILE_WIDTH_16    16
#define TILE_HEIGHT_16   16
#define TILE_WIDTH_32    32
#define TILE_HEIGHT_32   32

// Ej 1a) Kernel 
__global__ void transpose_global_kernel(float* d_input, int width, int height, float* d_output) {
    unsigned int imgx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int imgy = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (imgx < width && imgy < height) {
        d_output[(imgx*height) + imgy] = d_input[(imgy*width) + imgx];
    }
}

// Ej 1b) Kernel
__global__ void transpose_shared_kernel_16(float* d_input, int width, int height, float* d_output) {
    
    __shared__ float tile[TILE_WIDTH_16][TILE_HEIGHT_16];

    // Indices (x,y) en imagen de entrada
    unsigned int imgx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int imgy = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Lectura por fila (global) / Escritura por fila (compartida)
    if (imgx < width && imgy < height ) {
        tile[threadIdx.y][threadIdx.x] = d_input[(imgy*width) + imgx];
        __syncthreads();
    }

    // Indices (x,y) en imagen de salida ((y,x) en imagen de entrada)
    imgx = (blockIdx.y * blockDim.x) + threadIdx.x;
    imgy = (blockIdx.x * blockDim.y) + threadIdx.y;

    // Lectura por columna (compartida) / Escritura por fila (global)
    if (imgx < height && imgy < width) {
        d_output[(imgy*height) + imgx] = tile[threadIdx.x][threadIdx.y] ;
    }
}
__global__ void transpose_shared_kernel_32(float* d_input, int width, int height, float* d_output) {
    
    __shared__ float tile[TILE_WIDTH_32][TILE_HEIGHT_32];

    // Indices (x,y) en imagen de entrada
    unsigned int imgx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int imgy = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Lectura por fila (global) / Escritura por fila (compartida)
    if (imgx < width && imgy < height ) {
        tile[threadIdx.y][threadIdx.x] = d_input[(imgy*width) + imgx];
        __syncthreads();
    }

    // Indices (x,y) en imagen de salida ((y,x) en imagen de entrada)
    imgx = (blockIdx.y * blockDim.x) + threadIdx.x;
    imgy = (blockIdx.x * blockDim.y) + threadIdx.y;

    // Lectura por columna (compartida) / Escritura por fila (global)
    if (imgx < height && imgy < width) {
        d_output[(imgy*height) + imgx] = tile[threadIdx.x][threadIdx.y] ;
    }
}

// Ej 1c) Kernel 
__global__ void transpose_shared_extra_kernel_16(float* d_input, int width, int height, float* d_output) {
    
    __shared__ float tile[TILE_WIDTH_16][TILE_HEIGHT_16 + 1];

    // Indices (x,y) en imagen de entrada
    unsigned int imgx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int imgy = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Lectura por fila (global) / Escritura por fila (compartida)
    if (imgx < width && imgy < height ) {
        tile[threadIdx.y][threadIdx.x] = d_input[(imgy*width) + imgx];
        __syncthreads();
    }

    // Indices (x,y) en imagen de salida ((y,x) en imagen de entrada)
    imgx = (blockIdx.y * blockDim.x) + threadIdx.x;
    imgy = (blockIdx.x * blockDim.y) + threadIdx.y;

    // Lectura por columna (compartida) / Escritura por fila (global)
    if (imgx < height && imgy < width) {
        d_output[(imgy*height) + imgx] = tile[threadIdx.x][threadIdx.y] ;
    }
}
__global__ void transpose_shared_extra_kernel_32(float* d_input, int width, int height, float* d_output) {
    
    __shared__ float tile[TILE_WIDTH_32][TILE_HEIGHT_32 + 1];

    // Indices (x,y) en imagen de entrada
    unsigned int imgx = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int imgy = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Lectura por fila (global) / Escritura por fila (compartida)
    if (imgx < width && imgy < height ) {
        tile[threadIdx.y][threadIdx.x] = d_input[(imgy*width) + imgx];
        __syncthreads();
    }

    // Indices (x,y) en imagen de salida ((y,x) en imagen de entrada)
    imgx = (blockIdx.y * blockDim.x) + threadIdx.x;
    imgy = (blockIdx.x * blockDim.y) + threadIdx.y;

    // Lectura por columna (compartida) / Escritura por fila (global)
    if (imgx < height && imgy < width) {
        d_output[(imgy*height) + imgx] = tile[threadIdx.x][threadIdx.y] ;
    }
}

void transpose_gpu(float * img_in, int width, int height, float * img_out, int block_size, int algorithm){
    
    switch(algorithm) {
        case 1:
            printf("\n");
            printf("-> Kernel con memoria global\n");
            break;
        case 2:
            printf("\n");
            if (block_size == 16)
                printf("-> Kernel con memoria compartida, tamaño de bloque = 16\n");
            else
                printf("-> Kernel con memoria compartida, tamaño de bloque = 32\n");
            break;
        case 3:
            printf("\n");    
            if (block_size == 16)
                printf("-> Kernel con memoria compartida y columna extra, tamaño de bloque = 16\n");
            else
                printf("-> Kernel con memoria compartida y columna extra, tamaño de bloque = 32\n");;
            printf("\n");
            break;
        default:
            printf("Invocar como: './ej1.x nombre_archivo, algoritmo, tamaño_bloque'\n");
            printf("-> Algoritmo:\n");
            printf("\t 1 - Kernel con memoria global\n");
            printf("\t 2 - Kernel con memoria compartida\n");
            printf("\t 3 - Kernel con memoria compartida y columna extra\n");
            printf("\t 0 - Todos los algoritmos\n");
            printf("-> Tamaño de bloque:\n");
            printf("\t 16\n");
            printf("\t 32\n");
    }

    // Auxiliar para contar tiempo total
    float t_total = 0;
    
    // Etapa 1: Reserva de Memoria
    CLK_CUEVTS_INIT;
    CLK_CUEVTS_START;
    // Reserva en CPU
    unsigned int size = width * height * sizeof(float);
    float * device_img_in = (float *)malloc(size);
    float * device_img_out = (float *)malloc(size);
    // Reserva en GPU
    CUDA_CHK(hipMalloc((void**)& device_img_in, size));
    CUDA_CHK(hipMalloc((void**)& device_img_out, size));
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion GPU (Reserva de memoria): %f ms\n", t_elap);
    t_total = t_total + t_elap;
    
    // Etapa 2: Transferencia de datos (Host -> Device)
    CLK_CUEVTS_START;
    CUDA_CHK(hipMemcpy(device_img_in, img_in, size, hipMemcpyHostToDevice)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion GPU (Transferencia de datos (Host -> Device)): %f ms\n", t_elap);
    t_total = t_total + t_elap;

    // Etapa 3: Definir grilla
    int block_amount_x = width / block_size + (width % block_size != 0); // Division with ceiling
    int block_amount_y = height / block_size + (height % block_size != 0); // Division with ceiling
    dim3 tamGrid(block_amount_x, block_amount_y); // Grid dimension
    dim3 tamBlock(block_size, block_size); // Block dimension

    // Etapa 4 : Lanzar Kernel
    CLK_CUEVTS_START;
    switch(algorithm) {
        case 1:
            transpose_global_kernel<<<tamGrid, tamBlock>>>(device_img_in, width, height, device_img_out);
            break;
        case 2:
            if (block_size == 16)
                transpose_shared_kernel_16<<<tamGrid, tamBlock>>>(device_img_in, width, height, device_img_out);
            else
                transpose_shared_kernel_32<<<tamGrid, tamBlock>>>(device_img_in, width, height, device_img_out);
            break;
        case 3:
            if (block_size == 16)
                transpose_shared_extra_kernel_16<<<tamGrid, tamBlock>>>(device_img_in, width, height, device_img_out);
            else
                transpose_shared_extra_kernel_32<<<tamGrid, tamBlock>>>(device_img_in, width, height, device_img_out);
            break;
        default:
            transpose_global_kernel<<<tamGrid, tamBlock>>>(device_img_in, width, height, device_img_out);
    }
    // Sincronizar threads antes de parar timers
    hipDeviceSynchronize(); 
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion GPU (Kernel): %f ms\n", t_elap);
    t_total = t_total + t_elap;

    // Etapa 5: Transferencia de Datos (Device -> Host)
    CLK_CUEVTS_START;
    CUDA_CHK(hipMemcpy(img_out, device_img_out, size, hipMemcpyDeviceToHost)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia
    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;
    printf("Tiempo transposicion GPU (Transferencia de datos (Host <- Device)): %f ms\n", t_elap);
    t_total = t_total + t_elap;
    printf("Tiempo transposicion GPU: %f ms\n", t_total);
    printf("\n");

    // Etapa 6: Liberación de Memoria
    CUDA_CHK(hipFree(device_img_in));
    CUDA_CHK(hipFree(device_img_out));
}
