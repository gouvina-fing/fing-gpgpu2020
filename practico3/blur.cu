#include "hip/hip_runtime.h"
#include "util.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <algorithm>    // std::min std::max

using namespace std;

// CUDA Thread Indexing Cheatsheet https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf

// Ej 2a) Kernel que aplica el filtro Gaussiano en la GPU
// Ejemplo filtro https://www.nvidia.com/content/nvision2008/tech_presentations/Game_Developer_Track/NVISION08-Image_Processing_and_Video_with_CUDA.pdf
// Ejemplo multiplicacion de matrices http://selkie.macalester.edu/csinparallel/modules/GPUProgramming/build/html/CUDA2D/CUDA2D.html
__global__ void blur_kernel(float* d_input, int width, int height, float* d_output, float* d_msk, int m_size) {
    
    // __shared__ float block_memory[1024];

    int imgx = (blockIdx.x * blockDim.x) + threadIdx.x;
    imgx = max(0, imgx);
    imgx = min(imgx, width - 1);

    int imgy = (blockIdx.y * blockDim.y) + threadIdx.y;
    imgy = max(0, imgy);
    imgy = min(imgy, height - 1);

    //int block_index = (threadIdx.y * blockDim.y) + threadIdx.x;
    //block_memory[block_index] = d_input[(imgy*width) + imgx];
    
    __syncthreads();

    float val_pixel = 0;

    // Aca aplicamos la mascara
    for (int i = 0; i < m_size; i++) {
        for (int j = 0; j < m_size; j++) {
            
            int i2 = i - m_size/2;
            int j2 = j - m_size/2;

            int ix = imgx + i2;
            int iy = imgy + j2;
            
            //int bindex = ((threadIdx.y + i2) * blockDim.y) + (threadIdx.x + j2);

            // Altera el valor de un pixel, según sus vecinos.
            if (ix >= 0 && ix < width && iy >= 0 && iy < height) { // && bindex >= 0 && bindex < 1024) {
                //val_pixel = val_pixel +  block_memory[bindex] * d_msk[i*m_size+j];
                val_pixel = val_pixel +  d_input[(iy*width) + ix] * d_msk[i*m_size+j];
            }
        }
    }

    
    if (imgx < width && imgy < height) {
        d_output[(imgy*width) + imgx] = min(255.0f, max(0.0f, val_pixel));
    }
}

// Ej 1a) Threads con índice consecutivo en la dirección x deben acceder a pixels de una misma fila de la imagen.
//        Es importante usar blockIdx.x, blockIdx.y, threadIdx.x y threadIdx.y adecuadamente para acceder a la estructura bidimensional.
__global__ void ajustar_brillo_coalesced_kernel(float* d_input, float* d_output, int width, int height, float coef) {
    int imgx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int imgy = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (imgx < width && imgy < height) {
        d_output[(imgy*width) + imgx] = min(255.0f, max(0.0f, d_input[(imgy*width) + imgx] + coef));
    }
}

// Ej 1a) Threads con índice consecutivo en la dirección x deben acceder a pixels de una misma columna de la imagen.
//        Es importante usar blockIdx y threadIdx adecuadamente para acceder a la estructura bidimensional.
__global__ void ajustar_brillo_no_coalesced_kernel(float* d_input, float* d_output, int width, int height, float coef) {
    int imgx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int imgy = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (imgx < width && imgy < height) {
        d_output[(imgx*height) + imgy] = min(255.0f, max(0.0f, d_input[(imgx*height) + imgy] + coef));
    }
}

// Procesa la img en GPU sumando un coeficiente entre -255 y 255 a cada píxel, aumentando o reduciendo su brillo.
void ajustar_brillo_gpu(float * img_in, int width, int height, float * img_out, float coef, int algorithm, int filas=1) {

    // Tamaño de img_in en memoria
    unsigned int size = width * height * sizeof(float);
    float * device_img_in = (float *)malloc(size);
    float * device_img_out = (float *)malloc(size);

    // Reservo memoria en la GPU
    CUDA_CHK(hipMalloc((void**)& device_img_in, size));
    CUDA_CHK(hipMalloc((void**)& device_img_out, size));

    // Copio los datos a la memoria de la GPU
    CUDA_CHK(hipMemcpy(device_img_in, img_in, size, hipMemcpyHostToDevice)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia
   
    // Configurar grilla y lanzar kernel
    // TODO: La grilla (bidimensional) de threads debe estar configurada para aceptar matrices de cualquier tamaño.
    int block_size = 32;
    int block_amount_x = width / block_size + (width % block_size != 0); // Division with ceiling
    int block_amount_y = height / block_size + (height % block_size != 0); // Division with ceiling

    dim3 tamGrid(block_amount_x, block_amount_y); // Grid dimension
    dim3 tamBlock(block_size, block_size); // Block dimension

    // Lanzar kernel
    CLK_CUEVTS_INIT;
    CLK_CUEVTS_START;

    switch(algorithm) {
        case 1:
            ajustar_brillo_coalesced_kernel<<<tamGrid, tamBlock>>>(device_img_in, device_img_out, width, height, coef);
            break;
        case 2:
            ajustar_brillo_no_coalesced_kernel<<<tamGrid, tamBlock>>>(device_img_in, device_img_out, width, height, coef);
            break;
    }
    hipDeviceSynchronize();

    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;

    printf("Tiempo ajustar brillo GPU: %f ms\n", t_elap);

    // Transferir resultado a la memoria principal
    CUDA_CHK(hipMemcpy(img_out, device_img_out, size, hipMemcpyDeviceToHost)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia

    // TODO: Ej 1b) Registrar tiempos de cada etapa de ajustar_brillo_gpu para las dos variantes. Discutir diferencia entre variantes.
    //              (tiempos, reserva de memoria, transferencia de datos, ejecución del kernel, etc)
    //              Usar ambos mecanismo de medidas de utils.h (deberian dar casi igual)

    // TODO: Ej 1c) Compare los resultados de la salidad de nvprof.
    //              Registrar con nvprof --profileapi-trace none --metrics gld_efficiency ./blur imagen.ppm
    //              Qué puede decir del resultado de la métrica gld_efficiency?
    //              Duda: Esto se hace acá o en main.cpp?

    // Libero la memoria en la GPU
    CUDA_CHK(hipFree(device_img_in));
    CUDA_CHK(hipFree(device_img_out));
}

// Ej 2) Aplica un filtro Gaussiano que reduce el ruido de una imagen en escala de grises.
//       El filtro sustituye el valor de intensidad de cada pixel por un promedio ponderado de los pixeles vecinos.
//       Los pesos por los cuales se pondera cada vecino en el promedio se almacenan en una matriz cuadrada (máscara)
void blur_gpu(float * img_in, int width, int height, float * img_out, float msk[], int m_size){
    
    // Tamaño de img_in en memoria
    unsigned int size = width * height * sizeof(float);
    float * device_img_in = (float *)malloc(size);
    float * device_img_out = (float *)malloc(size);

    // Reservo memoria en la GPU
    CUDA_CHK(hipMalloc((void**)& device_img_in, size));
    CUDA_CHK(hipMalloc((void**)& device_img_out, size));

    // Copio los datos a la memoria de la GPU
    CUDA_CHK(hipMemcpy(device_img_in, img_in, size, hipMemcpyHostToDevice)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia
   
    // Configurar grilla y lanzar kernel
    // TODO: La grilla (bidimensional) de threads debe estar configurada para aceptar matrices de cualquier tamaño.
    int block_size = 32;
    int block_amount_x = width / block_size + (width % block_size != 0); // Division with ceiling
    int block_amount_y = height / block_size + (height % block_size != 0); // Division with ceiling

    dim3 tamGrid(block_amount_x, block_amount_y); // Grid dimension
    dim3 tamBlock(block_size, block_size); // Block dimension

    CLK_CUEVTS_INIT;
    CLK_CUEVTS_START;

    blur_kernel<<<tamGrid, tamBlock>>>(device_img_in, width, height, device_img_out, msk, m_size);
    hipDeviceSynchronize();

    CLK_CUEVTS_STOP;
    CLK_CUEVTS_ELAPSED;

    printf("Tiempo filtro gaussiano GPU: %f ms\n", t_elap);

    // Transferir resultado a la memoria principal
    CUDA_CHK(hipMemcpy(img_out, device_img_out, size, hipMemcpyDeviceToHost)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia

    // TODO: Ej 2b) Registre los tiempos de cada etapa de la función y compare las variantes de CPU y GPU.
    //              Usar ambos mecanismo de medidas de utils.h (deberian dar casi igual)
    //              ¿Qué aceleración se logra? ¿Y considerando únicamente el tiempo del kernel (hipMemcpy tiene mucho overhead!)?
    //              Duda: Esto se hace acá o en main.cpp?

    // Libero la memoria en la GPU
    CUDA_CHK(hipFree(device_img_in));
    CUDA_CHK(hipFree(device_img_out));
}

// Recorre la imagen sumando secuencialmente un coeficiente entre -255 y 255 a cada píxel, aumentando o reduciendo su brillo.
void ajustar_brillo_cpu(float * img_in, int width, int height, float * img_out, float coef) {

    CLK_POSIX_INIT;
    CLK_POSIX_START;

    for(int imgx=0; imgx < width ; imgx++) {
        for(int imgy=0; imgy < height; imgy++) {
            img_out[imgy*width+imgx] = min(255.0f,max(0.0f,img_in[imgy*width+imgx]+coef));
        }
    }

    CLK_POSIX_STOP;
    CLK_POSIX_ELAPSED;

    printf("Tiempo ajustar brillo CPU: %f ms\n", t_elap);
}

// Recorre la imagen aplicando secuencialmente un filtro Gaussiano que reduce el ruido de una imagen en escala de grises.
void blur_cpu(float * img_in, int width, int height, float * img_out, float msk[], int m_size) {

    CLK_POSIX_INIT;
    CLK_POSIX_START;

    float val_pixel=0;
    
    // Para cada pixel aplicamos el filtro
    for(int imgx=0; imgx < width ; imgx++) {
        for(int imgy=0; imgy < height; imgy++) {

            val_pixel = 0;

            // Aca aplicamos la mascara
            for (int i = 0; i < m_size ; i++) {
                for (int j = 0; j < m_size ; j++) {
                    
                    int ix =imgx + i - m_size/2;
                    int iy =imgy + j - m_size/2;
                    
                    // Altera el valor de un pixel, según sus vecinos.
                    if(ix >= 0 && ix < width && iy>= 0 && iy < height)
                        val_pixel = val_pixel +  img_in[iy * width +ix] * msk[i*m_size+j];
                }
            }
            
            // Guardo valor resultado
            img_out[imgy*width+imgx]= val_pixel;
        }
    }

    CLK_POSIX_STOP;
    CLK_POSIX_ELAPSED;

    printf("Tiempo filtro Gaussiano CPU: %f ms\n", t_elap);
}