#include "hip/hip_runtime.h"
#include "util.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"
#include <algorithm>    // std::min std::max

using namespace std;

#define TILE_WIDTH   32
#define TILE_HEIGHT  32
// TODO: Definir estos para cargar la shared memory (Cambiarles el nombre?)
#define BLOCK_WIDTH  TILE_WIDTH     // Width of shared memory block
#define BLOCK_HEIGHT TILE_HEIGHT    // Height of shared memory block

// Resolución de ecuaciones matriciales. Usando doble presición
// A × X = αB, donde α es un escalar, X y B ∈ R^{m×n}, y A ∈ R^{m×m} es una matriz triangular (inferior para esta implementación).
// Esto equivale a resolver n sistemas de ecuaciones de forma Ax_i = b_i, donde b_i es una columna de B y x_i es la solución buscada
// Al ser la matriz triangular el sistema de ecuaciones lineales ya viene "escalerizado".

// Ej 2.1 a) Caso 32 x n
// Para resolver estos sistemas:
//      - Cada bloque de threads debe sobreescribir un tile de B con el resultado de la operación.
//      - Cada warp del bloque procesa una columna de 32 elementos (resuelve uno de los n sistemas de ecuaciones). Como todos usan A hay que guardarla en memoria rápida.
//      - Cada thread del warp calcula un elemento de la columna().
//      - Cada thread lee datos calculados por los threads del warp del índice anterior. Para compartir datos entre los hilos del warp tenemos las siguientes opciones:

// Ej 2.1 a-1) Kernel para el caso 32 x n con los threads de un warp comunicandose a través memoria compartida
    // El paralelismo a nivel de warps es implicito, porque dentro de un warp se avanza en el código secuencialmente

__global__ void dtrsm_32_shared_kernel(const double alpha, double *d_A, int lda, double *d_B, int ldb, int stride_A, int stride_B) {
    __shared__ double shared_A[TILE_WIDTH][TILE_HEIGHT];
    __shared__ double tile_B[TILE_WIDTH][TILE_HEIGHT];

    double aux;
    int x, y, row_b, memory_index_x, memory_index_y;

    x = (blockIdx.x * blockDim.x) + threadIdx.x; // Column
    y = (blockIdx.y * blockDim.y) + threadIdx.y; // Row
    memory_index_x = threadIdx.x;
    memory_index_y = threadIdx.y;
    row_b = y*ldb;

    // Cada bloque guarda su pixel de A en memoria compartida
    shared_A[memory_index_y][memory_index_x] = d_A[memory_index_y*lda + memory_index_x + stride_A];
    aux = alpha*d_B[row_b + x + stride_B];
    __syncthreads();

    for(int k = 0; k <= memory_index_y; ++k) {
        if(k == memory_index_y) {
            // Se llegó a la diagonal de A, la incógnita queda resuelta y se guarda su resultado
            tile_B[k][memory_index_x] = aux/shared_A[k][k];
        } else {
            // Se va acumulando la resta de productos mientras se sube por la diagonal de A.
            aux -= shared_A[memory_index_y][k]*tile_B[k][memory_index_x];
        }
    }
    d_B[row_b + x + stride_B] = tile_B[memory_index_y][memory_index_x];
}

// Ej 2.1 a-2) Kernel para el caso 32 x n con los threads de un warp comunicandose utilizando la primitiva __shfl_sync
__global__ void dtrsm_32_shuffle_kernel(const double alpha, double *d_A, int lda, double *d_B, int ldb, int stride_A, int stride_B) {
    __shared__ double shared_A[TILE_WIDTH][TILE_HEIGHT];

    int x, y, row_a, row_b, memory_index_x, memory_index_y;
    double result, aux;

    x = (blockIdx.x * blockDim.x) + threadIdx.x; // Column
    y = (blockIdx.y * blockDim.y) + threadIdx.y; // Row
    memory_index_x = threadIdx.x;
    memory_index_y = threadIdx.y;
    row_b = y*ldb;

    // Cada bloque guarda su pixel de A en memoria compartida
    shared_A[memory_index_y][memory_index_x] = d_A[memory_index_y*lda + memory_index_x + stride_A];
    aux = alpha*d_B[row_b + x + stride_B];
    
    __syncthreads();

    // Los hilos de la fila 0 resuelven su incógnita, el resto adelanta la solución parcial de la misma.
    result = alpha*d_B[row_b + x]/shared_A[memory_index_y][memory_index_y];
    aux = __shfl_sync(0xffffffff, result, 0);

    __syncthreads();

    /*for(int k = 0; k < memory_index_y; ++k) {
        result -= shared_A[memory_index_y][k]*__shfl_sync(0xffffffff, result, k)/shared_A[memory_index_y][memory_index_y];
    }*/

    // Se itera por cada incógnita ya resuelta, usando su valor para resolver la siguiente y el resto parcialmente
    for(int k = 0; k < TILE_HEIGHT; ++k) {
        if(k < memory_index_y) {
            result -= shared_A[memory_index_y][k]*aux/shared_A[memory_index_y][memory_index_y];
        }
        aux = __shfl_sync(0xffffffff, result, k+1);
    }

    d_B[row_b + x + stride_B] = result;
}

__global__ void dgemm_shared_kernel(int p, const double alpha, double *d_A, int lda, double *d_B, int ldb, double beta, double *d_C, int ldc, int stride_A, int stride_B, int stride_C) {
    __shared__ double tile_A[TILE_WIDTH][TILE_HEIGHT];
    __shared__ double tile_B[TILE_WIDTH][TILE_HEIGHT];

    int x, y, k, row_a, row_c, memory_index_x, memory_index_y, idx, idy;
    double alpha_a, result;

    x = (blockIdx.x * blockDim.x) + threadIdx.x; // Column
    y = (blockIdx.y * blockDim.y) + threadIdx.y; // Row
    row_a = y*lda;
    row_c = y*ldc;
    result = d_C[row_c + x + stride_C]*beta;

    memory_index_x = threadIdx.x;
    memory_index_y = threadIdx.y;

    // Iteramos por cada bloque en las filas de A y columnas de B
    for(int step = 0; step < p; step+=32) {
        idx = step + memory_index_x;
        idy = step + memory_index_y;

        // Los hilos guardan el bloque en memoria compartida
        tile_A[memory_index_y][memory_index_x] = d_A[row_a + idx + stride_A];
        tile_B[memory_index_y][memory_index_x] = d_B[idy*ldb + x + stride_B];
        __syncthreads();

        // Se opera acediendo a los bloques previamente guardados
        for(k = 0; k < 32; ++k) {
            alpha_a = alpha*tile_A[memory_index_y][k];
            result += alpha_a*tile_B[k][memory_index_x];
        }
        // Se sincroniza para evitar que la memoria compartida sea editada mientras aún se usa para operar
        __syncthreads();
    }

    d_C[row_c + x + stride_C] = result;
}


// Ej 2.2) Función para el caso 32k x n con los threads de un warp comunicandose a través de la mejor variante de 2.1
// Acá la matriz triangular es de 32k x 32k, y podemos dividirla en k x k tiles de 32 x 32 elementos. Con:
//      - Tiles diagonales (matrices triangulares)
//      - Tiles no diagonales (matrices que no poseen estructura triangular)
// Para resolver n sistemas de 32k:
//      - Cada bloque de threads procesará 32 columnasde B (Recorriendo los tiles de A_{i,j} secuencialmente de izq a der y arriba hacia abajo)
//          Si el tile es diagonal la operacion es idéntica al caso anterior.
//          Si el tile no es diagonal la operación a realizar es la actualización del tile B_{i} mediante una operación DGEMM con tiles de 32x32
//              NOTE: Ver Figura 5. Observar que una operación muy similar es realizada como parte del procedimiento por tiles de la operación DGEMM de la parte anterior.
// NOTE: El parlaelismo grande está en la matriz B (cada fila de bloquecito en B se resuelve en paralelo). Pero las recorridas por los bloques de A son seriales
// Hay que recorrer secuencial en A porque es el orden que te impone la operación
void dtrsm_32k(int block_amount_x, int block_amount_y, const double alpha, double *d_A, int lda, double *d_B, int ldb, int meta_stride_A, int meta_stride_B) {
    // A es de 32k x 32k. En donde k == block_amount_x
    // B es de 32k x n. En donde k == block_amount_x y n = 32*block_amount_y

    int stride_A, stride_B, stride_C;
    dim3 tamGrid(1, block_amount_y); // Grid dimension
    dim3 tamBlock(TILE_WIDTH, TILE_HEIGHT); // Block dimension

    for(int i = 0; i < block_amount_x; ++i) {
        stride_A = meta_stride_A + 32*i*lda; // Move the stride in A to the next block of rows.
        stride_B = meta_stride_B + 32*(i-1)*ldb; // Move the stride in B to the previous block of rows (Not used when i = 0).
        stride_C = meta_stride_B + stride_B + 32*ldb; // Move the stride in C to the next block of rows.
        for(int j = 0; j <= i; ++j) {
            if (i == j) { // Diagonal
                dtrsm_32_shared_kernel<<<tamGrid, tamBlock>>>(alpha, d_A, lda, d_B, ldb, stride_A, stride_C);
            } else { // No diagonal
                // Bi = Bi - Aij * Bj
                // Bi = 32 x n (fila superior). Bj = 32 x n (fila inferior a actualizar). A = 32 x 32. p == n
                dgemm_shared_kernel<<<tamGrid, tamBlock>>>(32*block_amount_y, -1.0, d_A, lda, d_B, ldb, 1.0, d_B, ldb, stride_A, stride_B, stride_C);
            }
            stride_A += 32; // Move the stride in A to the next column block
        }
    }
}


// Ej 3.3) Kernel que implementa una solución recursiva de DTRSM empleando DGEMM y dividiendo la matriz triangular en tiles de 32x32. 
//         El paso base es DTRSM 32 x n ó DTRSM 32k x n (para un k pequeño) (TODO: Elegir viendo Figura 6 y video. NOTE: No es necesario usar k=32, podemos usar algo más chico)
//         El paso recursivo divide la matriz A en 4 submatrices (Y a B de forma coherente).
// NOTE: Ver letra y Figura 6 para las operaciones con las submatrices
//       Puede ser implementada en CPU (invocando los kernels correspondientes en cada caso, así es moar sencillo)
// NOTE: No es obligatorio experimentar con muchos valores de K.
void dtrsm_recursive(int m, int block_amount_y, const double alpha, double *d_A, int lda, double *d_B, int ldb, int stride_A, int stride_B) {
    if(m == 64) { // Paso base, A 32*2 x 32*2
        dtrsm_32k(2, block_amount_y, alpha, d_A, lda, d_B, ldb, stride_A, stride_B);
    } else { // Paso recursivo
        // A y B se parten en: |A11  0 |  |B1|
        //                     |A21 A22|  |B2|

        m = m/2;
        dim3 tamGrid(m/32, block_amount_y); // Grid dimension
        dim3 tamBlock(TILE_WIDTH, TILE_HEIGHT); // Block dimension
        
        // Se procesa A11, manteniendo direcciones de memoria.
        dtrsm_recursive(m, block_amount_y, alpha, d_A, lda, d_B, ldb, stride_A, stride_B);

        // Se procesa A21 (DGEMM), shifteando las direcciones de memoria al bloque de filas de abajo.
        dgemm_shared_kernel<<<tamGrid, tamBlock>>>(32*block_amount_y, -1.0, d_A, lda, d_B, ldb, 1.0, d_B, ldb, stride_A + m*lda, stride_B, stride_B + m*ldb);

        // Se procesa A22, shifteando las direcciones de memoria al bloque de filas de abajo y A m columnas hacia la derecha.
        dtrsm_recursive(m, block_amount_y, alpha, d_A, lda, d_B, ldb, stride_A + m*lda + m, stride_B + m*ldb);
    }
}

// A y B son arreglos unidimensionales de m × lda y n × ldb elementos.
// Para A el triángulo inferior del bloque superior izquierdo de tamaño m×m debe contener a A en su totalidad (El triangulo superior no es referenciado)
//
// La operación es in-place (los resultados se devuelven en la matriz B)
// TODO: En CuBlas alpha es un double *
void dtrsm_gpu(int algorithm, int m, int n, const double alpha, double *A, int lda, double *B, int ldb) {
    // Etapa 1: Reserva de Memoria
    unsigned int size_a = m*lda*sizeof(double);
    unsigned int size_b = ldb*n*sizeof(double);

    // Reserva en CPU
    double * device_A = (double *)malloc(size_a);
    double * device_B = (double *)malloc(size_b);
    
    // Reserva en GPU
    CUDA_CHK(hipMalloc((void**)& device_A, size_a));
    CUDA_CHK(hipMalloc((void**)& device_B, size_b));

    // Etapa 2: Transferencia de datos (Host -> Device)
    CUDA_CHK(hipMemcpy(device_A, A, size_a, hipMemcpyHostToDevice)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia
    CUDA_CHK(hipMemcpy(device_B, B, size_b, hipMemcpyHostToDevice));

    // Etapa 3: Definir grilla
    // Se crea una grilla con las dimensiones de B
    int block_amount_x = m / TILE_WIDTH + (m % TILE_WIDTH != 0); // Division with ceiling
    int block_amount_y = n / TILE_HEIGHT + (n % TILE_HEIGHT != 0); // Division with ceiling
    dim3 tamGrid(block_amount_x, block_amount_y); // Grid dimension
    dim3 tamBlock(TILE_WIDTH, TILE_HEIGHT); // Block dimension

    // Etapa 4 : Lanzar Kernel
    switch(algorithm) {
        case 3: // Versión 32 x n
            dtrsm_32_shared_kernel<<<tamGrid, tamBlock>>>(alpha, device_A, lda, device_B, ldb, 0, 0);
            break;
        case 4: // Versión 32k x n
            dtrsm_32k(block_amount_x, block_amount_y, alpha, device_A, lda, device_B, ldb, 0, 0);
            break;
        case 5: // Versión recursiva.
            dtrsm_recursive(m, block_amount_y, alpha, device_A, lda, device_B, ldb, 0, 0);
            break;
        case 7: // Versión 32 x n Shuffle/Shared (la menos eficiente)
            dtrsm_32_shuffle_kernel<<<tamGrid, tamBlock>>>(alpha, device_A, lda, device_B, ldb, 0, 0);
    }
    hipDeviceSynchronize();

    // Etapa 5: Transferencia de Datos (Device -> Host)
    CUDA_CHK(hipMemcpy(B, device_B, size_b, hipMemcpyDeviceToHost));

    // Etapa 6: Liberación de Memoria
    CUDA_CHK(hipFree(device_A));
    CUDA_CHK(hipFree(device_B));
}

void dtrsm_cublas(int m, int n, const double *alpha, double *A, int lda, double *B, int ldb) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    hipblasHandle_t handle;

    // Etapa 1: Reserva de Memoria
    unsigned int size_a = m*lda*sizeof(double);
    unsigned int size_b = ldb*n*sizeof(double);

    // Reserva en CPU
    double * device_A = (double *)malloc(size_a);
    double * device_B = (double *)malloc(size_b);

    // Reserva en GPU
    CUDA_CHK(hipMalloc((void**)& device_A, size_a));
    CUDA_CHK(hipMalloc((void**)& device_B, size_b));

    // Etapa 2: Crear Handle de CuBlas
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return;
    }

    // Etapa 3: Transferencia de datos (Host -> Device)
    status = hipblasSetMatrix(m, m, sizeof(double), A, lda, device_A, lda);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download A failed\n");
        CUDA_CHK(hipFree(device_A));
        hipblasDestroy(handle);
        return;
    }
    status = hipblasSetMatrix (m, n, sizeof(double), B, ldb, device_B, ldb);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download B failed\n");
        CUDA_CHK(hipFree(device_A));
        CUDA_CHK(hipFree(device_B));
        hipblasDestroy(handle);
        return;
    }

    // Etapa 4 : Lanzar Kernel
    status = hipblasDtrsm(
        handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
        m, n, alpha, device_A, lda, device_B, ldb
    );
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("DTRSM operation failed\n");
        CUDA_CHK(hipFree(device_A));
        CUDA_CHK(hipFree(device_B));
        hipblasDestroy(handle);
        return;
    }

    // Etapa 5: Transferencia de Datos (Device -> Host)
    status = hipblasGetMatrix (m, n, sizeof(double), device_B, ldb, B, ldb);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed\n");
        hipblasDestroy(handle);
    }

    // Etapa 6: Liberación de Memoria
    CUDA_CHK(hipFree(device_A));
    CUDA_CHK(hipFree(device_B));
    //return EXIT_SUCCESS;
}