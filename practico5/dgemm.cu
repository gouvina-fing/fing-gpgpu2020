#include "hip/hip_runtime.h"
#include "util.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <algorithm>    // std::min std::max

using namespace std;

#define TILE_WIDTH   32
#define TILE_HEIGHT  32
// TODO:
#define BLOCK_WIDTH  TILE_WIDTH     // Width of shared memory block
#define BLOCK_HEIGHT TILE_HEIGHT    // Height of shared memory block

// Producto de matrices. Usando doble presición.
// C = βC + αA × B
// Si queremos restarle a C la multiplicación de Axb alpha se define como negativo

// lda, ldb y ldc tienen la cantidad de elementos por fila (width) de cada matriz (lda ≥ k, ldb ≥ n y ldc ≥ n)
// En gral A tiene tantas columnas como lda, B como ldb, etc.
// El sentido de los mismos es si queremos trabajar con submatrices.
// Ejemplo: A 1000x1000, B 1000x1000, C 100x100 C = A'*B' (Con A' y B' las sumatrices de 100x100 de arriba a la izq)
//          m = 100, n = 100, p = 100, lda = 1000, ldb = 1000, ldc = 100

// Link relevante: https://spatial-lang.org/gemm

// Ej 1a) Kernel
// Cada bloque calcula un tile de C, cada hilo un elemento de C.
// No emplea memoria compartida ni otras optimizaciones.
// Asumimos que los tamaños del tile siempre son multiplos del tamaño de bloque
__global__ void dgemm_global_kernel() {}

// Ej 1b) Kernel

// Cada bloque calcula un tile de C, cada hilo un elemento de C.
// Cada bloque va pasando tiles de A y B a memoria compartida, multiplicandolos, acumulando el resultado en un registro y luego cargando otros tiles de A y B.
// Asumimos que los tamaños del tile siempre son multiplos del tamaño de bloque
__global__ void dgemm_shared_kernel() {}

void dgemm_gpu(int algorithm, int m, int n, int p, double alpha, double *A, int lda, double *B, int ldb, double beta, double *C, int ldc) {
    // Etapa 1: Reserva de Memoria
    unsigned int size_a = m*p*sizeof(double);
    unsigned int size_b = p*n*sizeof(double);
    unsigned int size_c = m*n*sizeof(double);

    // Reserva en CPU
    double * device_A = (double *)malloc(size_a);
    double * device_B = (double *)malloc(size_b);
    double * device_C = (double *)malloc(size_c);
    
    // Reserva en GPU
    CUDA_CHK(hipMalloc((void**)& device_A, size_a));
    CUDA_CHK(hipMalloc((void**)& device_B, size_b));
    CUDA_CHK(hipMalloc((void**)& device_C, ssize_c));

    // Etapa 2: Transferencia de datos (Host -> Device)
    CUDA_CHK(hipMemcpy(device_A, A, size_a, hipMemcpyHostToDevice)); // puntero destino, puntero origen, numero de bytes a copiar, tipo de transferencia
    CUDA_CHK(hipMemcpy(device_B, B, size_b, hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy(device_C, C, size_c, hipMemcpyHostToDevice));

    // Etapa 3: Definir grilla
    // Se crea una grilla con las dimensiones de C (un hilo por pixel de C)
    int block_amount_x = m / TILE_WIDTH + (m % TILE_WIDTH != 0); // Division with ceiling
    int block_amount_y = n / TILE_HEIGHT + (n % TILE_HEIGHT != 0); // Division with ceiling
    dim3 tamGrid(block_amount_x, block_amount_y); // Grid dimension
    dim3 tamBlock(TILE_WIDTH, TILE_HEIGHT); // Block dimension

    // Etapa 4 : Lanzar Kernel
    switch(algorithm) {
        case 1:
            dgemm_global_kernel<<<tamGrid, tamBlock>>>(m, n, p, alpha, device_A, lda, device_B, ldb, beta, device_C, ldc);
            break;
        case 2:
            dgemm_shared_kernel<<<tamGrid, tamBlock>>>(m, n, p, alpha, device_A, lda, device_B, ldb, beta, device_C, ldc);
    }
    hipDeviceSynchronize();

    // Etapa 5: Transferencia de Datos (Device -> Host)
    CUDA_CHK(hipMemcpy(C, device_C, size_c, hipMemcpyDeviceToHost));

    // Etapa 6: Liberación de Memoria
    CUDA_CHK(hipFree(device_A));
    CUDA_CHK(hipFree(device_B));
    CUDA_CHK(hipFree(device_C));
}

void dgemm_cpu(int m, int n, int p, double alpha, double *A, int lda, double *B, int ldb, double beta, double *C, int ldc) {
    int i, j, k, row_a, row_b, row_c;
    double alpha_a;

    for(i = 0; i < m; ++i) {
        row_a = i*lda;
        row_c = i*ldc;
        for(j = 0; j < n; ++j)
            C[row_c + j] *= beta;

        for(k = 0; k < p; ++k) {
            row_b = k*ldb;
            alpha_a = alpha*A[row_a + k];
            for(j = 0; j < n; ++j)
                C[row_c + j] += alpha_a*B[row_b + j];
        }
    }
}